#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>

using namespace std;

#define GRID_SIZE 16
#define BLOCK_SIZE 16
#define NUM_LISTS 16
unsigned long const NUM_ELEMENT = (1 << 10);

template <class T>
void c_swap(T &x, T &y) {
  T tmp1 = x;
  x = y;
  y = tmp1;
}

__device__ void radix_sort(unsigned long *const sort_tmp,
                           unsigned long *const sort_tmp_1,
                           unsigned long *const sort_tmp11) {
  for (unsigned long bit_mask = 1; bit_mask > 0; bit_mask <<= 1) {
    unsigned int base_cnt_0 = 0;
    unsigned int base_cnt_1 = 0;

    for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        if (sort_tmp[i + tid] & bit_mask) {
          sort_tmp_1[base_cnt_1 + tid] = sort_tmp[i + tid];
          base_cnt_1 += NUM_LISTS;
        } else {
          sort_tmp[base_cnt_0 + tid] = sort_tmp[i + tid];
          base_cnt_0 += NUM_LISTS;
        }
      }
    }

    for (unsigned long i = 0; i < base_cnt_1; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
      }
    }
    __syncthreads();
  }
}

__global__ void merge_block(unsigned long *const g_idata, unsigned long g_odata) {
  __shared__ float partialSum[8];
  unsigned long tmp[2];
  //确定索引
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  //传global memory数据到shared memory
  partialSum[tid] = g_idata[i];

  //传输同步
  __syncthreads();

  //在共享存储器中进行规约
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (tid < stride)
      partialSum[tid] = min(partialSum[tid + stride], partialSum[tid]);
    __syncthreads();
  }
  //将当前block的计算结果写回输出数组
  if (tid == 0) tmp[blockIdx.x] = partialSum[0];
  __syncthreads();
}


__device__ int search_index(unsigned long *const array_tmp, unsigned long val) {
  int left = 0;
  int right = NUM_ELEMENT - 1;
  while (left <= right) {
    int middle = (right + left) / 2;
    if (array_tmp[middle] > val) {
      right = middle - 1;

    } else if (array_tmp[middle] < val) {
      left = middle + 1;
    } else {
      return middle;
    }
  }
  return -1;
  //二分法找元素index
}
template <typename S>
__global__ void sort_index(S *sortarray, unsigned long *const array_tmp,
                           unsigned long *const data) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      data[tid + i] = search_index(array_tmp, sortarray[i + tid].key);
    }
  }
  __syncthreads();
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      array_tmp[data[tid + i]] = tid + i;
    }
  }
  __syncthreads();
}

template <typename S>
__global__ void sort_struct(unsigned long *const array_tmp, S *sortarray,
                            S *struct_tmp) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      struct_tmp[tid + i] = sortarray[array_tmp[tid + i]];
    }
  }
  __syncthreads();

  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      sortarray[tid + i] = struct_tmp[tid + i];
    }
  }
  __syncthreads();
}

template <typename S>
__device__ int search(S *nums, unsigned long val) {
  for (int i = 0; i < NUM_ELEMENT; i++) {
    if (nums[i].key == val) {
      return i;
    }
  }
  return -1;
}
typedef struct SORTSTRUCT {
  unsigned long key;
} sorta;

sorta sortarray[NUM_ELEMENT];  //定义为全局变量避免堆栈溢出

int main(void) {
  for (unsigned long i = 0; i < NUM_ELEMENT; i++) {
    sortarray[i].key = i;
    // sortarray[i].key = i%35;//key值相等的情况
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    c_swap(sortarray[rand() % 7].key, sortarray[i].key);
  }
  sorta *gpu_sortarray;
  sorta *struct_tmp;
  unsigned long *array_tmp;
  unsigned long *array_tmp_2;
  unsigned long *sort_tmp;
  unsigned long *min_array;
  unsigned long *min_value;
  hipMalloc((sorta **)&gpu_sortarray, sizeof(sorta) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp_2, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&sort_tmp, sizeof(unsigned long) * NUM_LISTS);
  hipMalloc((void **)&min_array, sizeof(unsigned long) * GRID_SIZE);
  hipMalloc((sorta **)&struct_tmp, sizeof(sorta) * NUM_ELEMENT);

  radix_sort<<<GRID_SIZE, BLOCK_SIZE>>>(gpu_sortarray, array_tmp, sort_tmp);

  for (int i = 0; i < NUM_ELEMENT; i++) {
    min_value = 0xFFFFFFF;
    merge_block<<<GRID_SIZE, BLOCK_SIZE>>>(sort_tmp, min_array);
    merge_final<<<1, BLOCK_SIZE>>>(min_array, min_value);
    for (int j = 0; j < NUM_LISTS; j++) {
      if (self_data[j] == min_value) {
        sortarray_tmp[i] = min_value;
        index[j] = index[j] + 1;
      }
      if (j + index[j] * NUM_LISTS < NUM_ELEMENT) {
        sort_tmp[j] = array_tmp[j + index[j] * NUM_LISTS];
      } else {
        sort_tmp[j] = 0xFFFFFFFF;
      }
    }
  }
  sort_index<<<GRID_SIZE, BLOCK_SIZE>>>(gpu_sortarray, array_tmp, array_tmp_2);
  sort_struct<<<GRID_SIZE, BLOCK_SIZE>>>(array_tmp, gpu_sortarray, struct_tmp);

  /*
    unsigned long *gpu_srcData;
    unsigned long min_data;
    unsigned int *index;
    hipMalloc((void **)&gpu_srcData, sizeof(unsigned long) * NUM_ELEMENT);

    hipMalloc((void **)&self_data, sizeof(unsigned long) * NUM_LISTS);
    hipMalloc((void **)&index, sizeof(unsigned int) * NUM_LISTS);
    hipMalloc((void **)&min_data, sizeof(unsigned long));
  */

  hipMemcpy(gpu_sortarray, sortarray, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyHostToDevice);

  // hipError_t error = hipGetLastError();
  dim3 grid(2);
  dim3 block(8);

  hipEvent_t start, stop;  //定义事件
  hipEventCreate(&start);  //起始时间
  hipEventCreate(&stop);   //结束时间

  hipEventRecord(start, 0);  //记录起始时间
  cspincuda<<<grid, block>>>(gpu_srcData, array_tmp, gpu_sortarray, struct_tmp,
                             self_data, min_data, index);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);  //执行完代码，记录结束时间

  hipEventSynchronize(stop);
  hipMemcpy(sortarray, array_tmp, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  hipFree(gpu_srcData);
  hipFree(array_tmp);
  hipFree(gpu_sortarray);
  hipFree(struct_tmp);

  printf("CUDA error: %s\n", hipGetErrorString(error));

  int result = 0;
  for (int i = 0; i < NUM_ELEMENT - 1; i++) {
    if (sortarray[i].key > sortarray[i + 1].key) {
      result++;
      // printf("%ld\n",sortarray[i].key);
      // printf("%ld\n",sortarray[i+1].key);
    }
    // printf("%ld\n",sortarray[i].key);
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    printf("%ld\n", sortarray[i].key);
  }

  printf("%ld\n", NUM_ELEMENT);
  printf("%d\n", result);
  if (result == 0) {
    printf("result is true.\n");
  } else {
    printf("result is false.\n");
  }
  float elapsedTime;  //计算总耗时，单位ms
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("%f\n", elapsedTime);
}
