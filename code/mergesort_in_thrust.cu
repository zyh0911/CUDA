#include "hip/hip_runtime.h"
//merge sort
#include<iostream>
#include<math.h>
#include<hip/hip_runtime.h>
#include"" 
#include<fstream>
#include <algorithm>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
using namespace std;
unsigned long const  NUM_ELEMENT=(1<<7);
template<class T> void c_swap(T &x, T &y){ T tmp1 = x; x = y; y = tmp1; }

typedef struct SORTSTRUCT{
    unsigned long key;
    } sorta;

bool seed_compare(SORTSTRUCT a, SORTSTRUCT b)
    {
        return a.key< b.key;
    };
    sorta sortarray[NUM_ELEMENT];
int main(void)
{
    
    
    for(unsigned long i = 0; i < NUM_ELEMENT; i++)  
    {
        sortarray[i].key = i;
    }  
    for(int i = 0; i < NUM_ELEMENT; i++)
    {
        c_swap(sortarray[rand()%7].key, sortarray[i].key);
    }
    clock_t start, end;
    hipMallocManaged((void **)&sortarray,sizeof(sortarray)*NUM_ELEMENT);
    start = clock();
    thrust::sort(sortarray,sortarray+NUM_ELEMENT,seed_compare);
    end = clock();

    hipError_t error = hipGetLastError();   
    printf("CUDA error: %s\n", hipGetErrorString(error));
    
    
    int result=0;
    for(int i=0;i<NUM_ELEMENT-1;i++)
    {
        if(sortarray[i].key>sortarray[i+1].key)
        {
            result++;
            //printf("%ld\n",sortarray[i].key);
            //printf("%ld\n",sortarray[i+1].key);
        }
        //printf("%ld\n",sortarray[i].key);
    }
    /*
    for(int i=0;i<NUM_ELEMENT;i++)
    {
        printf("%ld\n",sortarray[i].key);
    }
    */
    printf("%ld\n",NUM_ELEMENT);
    printf("%d\n",result);
    if(result==0)
    {
        printf("result is true.\n");
    }
    else
    {
        printf("result is false.\n");
    }
    printf("run time is %.8lf\n", (double)(end-start)/CLOCKS_PER_SEC);
    
}
