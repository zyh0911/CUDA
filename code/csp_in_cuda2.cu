#include "hip/hip_runtime.h"
// csp in cuda 为了解决多block的问题，计划采用原子操作lock的方法
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>

using namespace std;
#define GRID_SIZE 2
#define NUM_LISTS 128
unsigned long const NUM_ELEMENT = (1 << 8);

template <class T>
void c_swap(T &x, T &y) {
  T tmp1 = x;
  x = y;
  y = tmp1;
}

template <typename S>
__device__ void copy_index(S *sortarray, unsigned long *const data,
                           const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    data[tid + i] = sortarray[tid + i].key;
  }
  __syncthreads();
}

__device__ void radix_sort(unsigned long *const sort_tmp,
                           unsigned long *const sort_tmp_1,
                           const unsigned int tid) {
  for (unsigned long bit_mask = 1; bit_mask > 0; bit_mask <<= 1) {
    unsigned int base_cnt_0 = 0;
    unsigned int base_cnt_1 = 0;

    for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        if (sort_tmp[i + tid] & bit_mask) {
          sort_tmp_1[base_cnt_1 + tid] = sort_tmp[i + tid];
          base_cnt_1 += NUM_LISTS;
        } else {
          sort_tmp[base_cnt_0 + tid] = sort_tmp[i + tid];
          base_cnt_0 += NUM_LISTS;
        }
      }
    }

    for (unsigned long i = 0; i < base_cnt_1; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
      }
    }
    __syncthreads();
  }
}

__device__ void reduce(unsigned long *const g_idata,
                       unsigned long *const g_odata) {
  __shared__ float partialSum[NUM_LISTS / GRID_SIZE];

  //确定索引
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  //传global memory数据到shared memory
  partialSum[tid] = g_idata[i];

  //传输同步
  __syncthreads();

  //在共享存储器中进行规约
  for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
    if (tid < stride)
      partialSum[tid] = min(partialSum[tid + stride], partialSum[tid]);
    __syncthreads();
  }
  //将当前block的计算结果写回输出数组
  if (tid == 0) g_odata[blockIdx.x] = partialSum[0];
  
 /*
  // int i = threadIdx.x + blockIdx.x * blockDim.x;
  // printf("%d is %ld\n",i,g_idata[i]);
  for (int j = 0; j < GRID_SIZE; j++) {
    g_odata[j] = 0xFFFFFFFF;
    for (int i = 0; i < NUM_LISTS / GRID_SIZE; i++) {
      g_odata[j] = min(g_odata[j], g_idata[i + j * NUM_LISTS / GRID_SIZE]);
    }
    printf("%d is %ld\n", j, g_odata[j]);
  }
*/
  __syncthreads();
}

__device__ void merge(unsigned long *const data, unsigned long *const array_tmp,
                      const unsigned int tid)  //分块的归约合并
{
  unsigned int index[GRID_SIZE][NUM_LISTS / GRID_SIZE];
  unsigned long self_data[NUM_LISTS];

  unsigned long min_value;
  unsigned long min_data[GRID_SIZE];
  self_data[tid] = data[tid];
  index[blockIdx.x][threadIdx.x] = 0;
  __syncthreads();
  // printf("%d is %ld\n",tid,data[tid]);
  for (int i = 0; i < 2; i++) {
    min_value = 0xFFFFFFFF;
    // min_data[blockIdx.x]= 1145151;
    __syncthreads();
    //printf("%d is %ld\n",tid,self_data[tid]);
    reduce(self_data, min_data);
    /*
    if(i==1)
    {printf("%ld\n", min_data[0]);
    //printf("%d\n", min_data[1]);
    }
    */
    for (int j = 0; j < GRID_SIZE; j++) {
      min_value = min(min_value, min_data[j]);
    }
    //atomicMin(&min_value,min_data[blockIdx.x]);
    __syncthreads();
    
    array_tmp[i] = min_value;
    __syncthreads();
    if (min_data[blockIdx.x] == min_value) {
      if (self_data[tid] == min_value) {
       printf("%d\n",tid);
       printf("%d\n", blockIdx.x);
       printf("%d\n", threadIdx.x);
      index[blockIdx.x][threadIdx.x] = index[blockIdx.x][threadIdx.x] + 1;
      if (index[blockIdx.x][threadIdx.x] < NUM_ELEMENT / NUM_LISTS) {
        self_data[tid] =
            data[threadIdx.x + index[blockIdx.x][threadIdx.x] * NUM_LISTS];
        //printf("%d\n", tid);
        // printf("%d\n",data[threadIdx.x + index[blockIdx.x][threadIdx.x] *
        // NUM_LISTS]);
      } else {
        self_data[tid] = 0xFFFFFFFF;
      }
      }
      // printf("%d\n", threadIdx.x);
    }
   __threadfence();
    /*
        for (int j = 0; j < GRID_SIZE; j++) {
          if (min_data[j] == min_value) {
            for (int k = 0; k < NUM_LISTS / GRID_SIZE; k++) {
              if (self_data[k] == min_value) {
                index[j][k] = index[j][k] + 1;
                if (index[j][k]  < NUM_ELEMENT/NUM_LISTS) {
                  self_data[k] = data[k + index[j][k] * NUM_LISTS ];
                } else {
                  self_data[k] = 0xFFFFFFFF;
                }

              }
            }
          }
        }*/
  }
}
/*
    unsigned int index[NUM_LISTS];
    unsigned int self_data[NUM_LISTS];

    unsigned int min_data[GRID_SIZE];
    unsigned int min_tid[GRID_SIZE];

    unsigned int min_tid_new;
    unsigned int min_value;

    self_data[tid] = 0xFFFFFFFF;
    index[tid] = 0;
    __syncthreads();
    for (int j = 0; j < GRID_SIZE; j++) {
      min_data[j]= 0xFFFFFFFF;
      min_tid[j]= 0xFFFFFFFF;
    }
    min_value=0xFFFFFFFF;
    min_tid_new=0xFFFFFFFF;
    for (int i = 0; i < NUM_ELEMENT; i++) {

      unsigned int block_tid=threadIdx.x + threadIdx.y*blockDim.x;

        if (tid + index[tid] * NUM_LISTS < NUM_ELEMENT) {
          self_data[tid] = data[tid + index[tid] * NUM_LISTS];
        }
        else {
          self_data[tid] = 0xFFFFFFFF;
        }
      for(int j=0;j<GRID_SIZE;j++)
      {
        for(int k=0;k<NUM_LISTS/GRID_SIZE;k++)
        {
          min_data[j]=min(min_data[j], self_data[k]);
        }
        if (self_data[block_tid] == min_value) {
          min_tid[j] = min(min_tid[j], block_tid+j*NUM_LISTS/GRID_SIZE);
        }
      }
      for (int j = 0; j < GRID_SIZE; j++) {
      min_value=min(min_value,min_data[j]);
      }
      __syncthreads();
      for (int j = 0; j < GRID_SIZE; j++) {
      if (min_data[j] == min_value) {
        min_tid_new= min(min_tid_new,min_tid[j]);
      }}
      __syncthreads();
      if (tid== min_tid_new) {
        array_tmp[i] = min_value;
        index[tid] = index[tid] + 1;
      }
      __syncthreads();
    }
    */

__device__ int search_index(unsigned long *const array_tmp, unsigned long val) {
  int left = 0;
  int right = NUM_ELEMENT - 1;
  while (left <= right) {
    int middle = (right + left) / 2;
    if (array_tmp[middle] > val) {
      right = middle - 1;

    } else if (array_tmp[middle] < val) {
      left = middle + 1;
    } else {
      return middle;
    }
  }
  return -1;
  //二分法找元素index
}
template <typename S>
__device__ void sort_index(S *sortarray, unsigned long *const array_tmp,
                           unsigned long *const data, const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      data[tid + i] = search_index(array_tmp, sortarray[i + tid].key);
    }
  }
  __syncthreads();
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      array_tmp[data[tid + i]] = tid + i;
    }
  }
  __syncthreads();
}

template <typename S>
__device__ void sort_struct(unsigned long *const array_tmp, S *sortarray,
                            S *struct_tmp, const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      struct_tmp[tid + i] = sortarray[array_tmp[tid + i]];
    }
  }
  __syncthreads();

  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      sortarray[tid + i] = struct_tmp[tid + i];
    }
  }
  __syncthreads();
}

template <typename S>
__device__ int search(S *nums, unsigned long val) {
  for (int i = 0; i < NUM_ELEMENT; i++) {
    if (nums[i].key == val) {
      return i;
    }
  }
  return -1;
}
typedef struct SORTSTRUCT {
  unsigned long key;
} sorta;

__global__ void cspincuda(unsigned long *const data,
                          unsigned long *const array_tmp, sorta *sortarray,
                          sorta *struct_tmp) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  copy_index(sortarray, data, tid);  // step1:copy index
  radix_sort(data, array_tmp, tid);
  // merge(data, array_tmp, tid);
  // printf("%d is %ld\n",tid,data[tid]);
  merge(data, array_tmp, tid);

  // sort_index(sortarray, array_tmp, data, tid);         //
  // step2:sort_by_key sort_struct(array_tmp, sortarray, struct_tmp, tid);
  //  step3:sort array
}

sorta sortarray[NUM_ELEMENT];  //定义为全局变量避免堆栈溢出

int main(void) {
  for (unsigned long i = 0; i < NUM_ELEMENT; i++) {
    sortarray[i].key = i;
    // sortarray[i].key = i%35;//key值相等的情况
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    c_swap(sortarray[rand() % 7].key, sortarray[i].key);
  }

  unsigned long *gpu_srcData;
  unsigned long *array_tmp;

  sorta *gpu_sortarray;
  sorta *struct_tmp;

  hipMalloc((void **)&gpu_srcData, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((sorta **)&gpu_sortarray, sizeof(sorta) * NUM_ELEMENT);
  hipMalloc((sorta **)&struct_tmp, sizeof(sorta) * NUM_ELEMENT);

  hipMemcpy(gpu_sortarray, sortarray, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyHostToDevice);

  // hipError_t error = hipGetLastError();
  dim3 grid(GRID_SIZE);
  dim3 block(NUM_LISTS / GRID_SIZE);

  hipEvent_t start, stop;  //定义事件
  hipEventCreate(&start);  //起始时间
  hipEventCreate(&stop);   //结束时间

  hipEventRecord(start, 0);  //记录起始时间
  cspincuda<<<grid, block>>>(gpu_srcData, array_tmp, gpu_sortarray, struct_tmp);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);  //执行完代码，记录结束时间

  hipEventSynchronize(stop);
  hipMemcpy(sortarray, array_tmp, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  hipFree(gpu_srcData);
  hipFree(array_tmp);
  hipFree(gpu_sortarray);
  hipFree(struct_tmp);

  int result = 0;
  for (int i = 0; i < NUM_ELEMENT - 1; i++) {
    if (sortarray[i].key != sortarray[i + 1].key - 1) {
      result++;
      // printf("%ld\n",sortarray[i].key);
      // printf("%ld\n",sortarray[i+1].key);
    }
    // printf("%ld\n",sortarray[i].key);
  }
  /*
  for (int i = 0; i < NUM_ELEMENT; i++) {
    printf("%ld\n", sortarray[i].key);
  }
*/
  printf("CUDA error: %s\n", hipGetErrorString(error));
  printf("%ld\n", NUM_ELEMENT);
  printf("%d\n", result);
  if (result == 0) {
    printf("result is true.\n");
  } else {
    printf("result is false.\n");
  }
  float elapsedTime;  //计算总耗时，单位ms
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("%f\n", elapsedTime);
}
