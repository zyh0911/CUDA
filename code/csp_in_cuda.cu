#include "hip/hip_runtime.h"
// csp in cuda new
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>

using namespace std;

#define NUM_LISTS 16
#define NUM_GRIDS 1
unsigned long const NUM_ELEMENT = (1 << 6);

template <class T>
void c_swap(T &x, T &y) {
  T tmp1 = x;
  x = y;
  y = tmp1;
}

template <typename S>
__device__ void copy_index(S *sortarray, unsigned long *const data,
                           const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    data[tid + i] = sortarray[tid + i].key;
  }
  __syncthreads();
}

__device__ void radix_sort(unsigned long *const sort_tmp,
                           unsigned long *const sort_tmp_1,
                           const unsigned int tid) {
  for (unsigned long bit_mask = 1; bit_mask > 0; bit_mask <<= 1) {
    unsigned int base_cnt_0 = 0;
    unsigned int base_cnt_1 = 0;

    for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        if (sort_tmp[i + tid] & bit_mask) {
          sort_tmp_1[base_cnt_1 + tid] = sort_tmp[i + tid];
          base_cnt_1 += NUM_LISTS;
        } else {
          sort_tmp[base_cnt_0 + tid] = sort_tmp[i + tid];
          base_cnt_0 += NUM_LISTS;
        }
      }
    }

    for (unsigned long i = 0; i < base_cnt_1; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
      }
    }
    __syncthreads();
  }
}
//这里有问题，block之间无法同步，导致信息传输有问题
__device__ void merge(unsigned long *const data, unsigned long *const array_tmp,
                      const unsigned int tid,unsigned long *const self_data,unsigned long min_data
                      ) {
  //__shared__ 
  unsigned int index[NUM_LISTS];
  //__shared__
  //unsigned int min_data;
  //__shared__
  //unsigned int min_tid;
  index[tid] = 0;
  __threadfence();

  for (int i = 0; i < NUM_ELEMENT; i++) {
    //__shared__ unsigned int self_data[NUM_LISTS];
    self_data[tid] = 0xFFFFFFFF;
    min_data = 0xFFFFFFFF;
    
    

    if (tid + index[tid] * NUM_LISTS < NUM_ELEMENT) {
      self_data[tid] = data[tid + index[tid] * NUM_LISTS];
      __threadfence();
    } else {
      self_data[tid] = 0xFFFFFFFF;
      __threadfence();
    }
    
    __threadfence();
    
    for (int j = 0; j < NUM_LISTS; j++) {
      min_data = min(min_data, self_data[j]);
    }
    //atomicMin(&min_data,self_data[tid]);
    __syncthreads();
    //__threadfence();
    //printf("%ld\n",min_data);
    for(int j=0;j<NUM_LISTS;j++)
    {
    if (self_data[j] == min_data) 
    /*{
      min_tid = min(min_tid, tid);
    }
    if (tid == min_tid) */
    {
      array_tmp[i] = min_data;
      index[j] = index[j] + 1;
    }

    }
    __threadfence();
  }
}
__device__ int search_index(unsigned long *const array_tmp, unsigned long val) {
  int left = 0;
  int right = NUM_ELEMENT - 1;
  while (left <= right) {
    int middle = (right + left) / 2;
    if (array_tmp[middle] > val) {
      right = middle - 1;

    } else if (array_tmp[middle] < val) {
      left = middle + 1;
    } else {
      return middle;
    }
  }
  return -1;
  //二分法找元素index
}
template <typename S>
__device__ void sort_index(S *sortarray, unsigned long *const array_tmp,
                           unsigned long *const data, const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      data[tid + i] = search_index(array_tmp, sortarray[i + tid].key);
    }
  }
  __syncthreads();
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      array_tmp[data[tid + i]] = tid + i;
    }
  }
  __syncthreads();
}

template <typename S>
__device__ void sort_struct(unsigned long *const array_tmp, S *sortarray,
                            S *struct_tmp, const unsigned int tid) {
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      struct_tmp[tid + i] = sortarray[array_tmp[tid + i]];
    }
  }
  __syncthreads();

  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      sortarray[tid + i] = struct_tmp[tid + i];
    }
  }
  __syncthreads();
}

template <typename S>
__device__ int search(S *nums, unsigned long val) {
  for (int i = 0; i < NUM_ELEMENT; i++) {
    if (nums[i].key == val) {
      return i;
    }
  }
  return -1;
}
typedef struct SORTSTRUCT {
  unsigned long key;
} sorta;

__global__ void cspincuda(unsigned long *const data,
                          unsigned long *const array_tmp, sorta *sortarray,
                          sorta *struct_tmp,unsigned long *const self_data,
                          unsigned long min_data) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  copy_index(sortarray, data, tid);  // step1:copy index
  radix_sort(data, array_tmp, tid);
  merge(data, array_tmp, tid,self_data,min_data);
  //sort_index(sortarray, array_tmp, data, tid);         // step2:sort_by_key
  //sort_struct(array_tmp, sortarray, struct_tmp, tid);  // step3:sort array
}

sorta sortarray[NUM_ELEMENT];  //定义为全局变量避免堆栈溢出

int main(void) {
  for (unsigned long i = 0; i < NUM_ELEMENT; i++) {
    sortarray[i].key = i;
    // sortarray[i].key = i%35;//key值相等的情况
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    c_swap(sortarray[rand() % 7].key, sortarray[i].key);
  }

  unsigned long *gpu_srcData;
  unsigned long *array_tmp;
  sorta *gpu_sortarray;
  sorta *struct_tmp;
  unsigned long *self_data;
  unsigned long min_data;
  hipMalloc((void **)&gpu_srcData, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&self_data, sizeof(unsigned long) * NUM_LISTS);
  hipMalloc((void **)&min_data, sizeof(unsigned long));
  hipMalloc((sorta **)&gpu_sortarray, sizeof(sorta) * NUM_ELEMENT);
  hipMalloc((sorta **)&struct_tmp, sizeof(sorta) * NUM_ELEMENT);

  hipMemcpy(gpu_sortarray, sortarray, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyHostToDevice);

  // hipError_t error = hipGetLastError();
  dim3 grid(2);
  dim3 block(8);

  hipEvent_t start, stop;  //定义事件
  hipEventCreate(&start);  //起始时间
  hipEventCreate(&stop);   //结束时间

  hipEventRecord(start, 0);  //记录起始时间
  cspincuda<<<grid, block>>>(gpu_srcData, array_tmp, gpu_sortarray, struct_tmp,self_data,min_data);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);  //执行完代码，记录结束时间

  hipEventSynchronize(stop);
  hipMemcpy(sortarray, array_tmp, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  hipFree(gpu_srcData);
  hipFree(array_tmp);
  hipFree(gpu_sortarray);
  hipFree(struct_tmp);

  printf("CUDA error: %s\n", hipGetErrorString(error));

  int result = 0;
  for (int i = 0; i < NUM_ELEMENT - 1; i++) {
    if (sortarray[i].key > sortarray[i + 1].key) {
      result++;
      // printf("%ld\n",sortarray[i].key);
      // printf("%ld\n",sortarray[i+1].key);
    }
    // printf("%ld\n",sortarray[i].key);
  }
  
  for(int i=0;i<NUM_ELEMENT;i++)
  {
      printf("%ld\n",sortarray[i].key);
  }
  
  printf("%ld\n", NUM_ELEMENT);
  printf("%d\n", result);
  if (result == 0) {
    printf("result is true.\n");
  } else {
    printf("result is false.\n");
  }
  float elapsedTime;  //计算总耗时，单位ms
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("%f\n", elapsedTime);
}
