#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <fstream>
#include <iostream>

using namespace std;

#define GRID_SIZE 16
#define BLOCK_SIZE 16
#define NUM_LISTS 16
unsigned long const NUM_ELEMENT = (1 << 10);

template <class T>
void c_swap(T &x, T &y) {
  T tmp1 = x;
  x = y;
  y = tmp1;
}

__device__ void radix_sort(unsigned long *const sort_tmp,
                           unsigned long *const sort_tmp_1,
                           unsigned long *const sort_tmp11) {
  for (unsigned long bit_mask = 1; bit_mask > 0; bit_mask <<= 1) {
    unsigned int base_cnt_0 = 0;
    unsigned int base_cnt_1 = 0;

    for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        if (sort_tmp[i + tid] & bit_mask) {
          sort_tmp_1[base_cnt_1 + tid] = sort_tmp[i + tid];
          base_cnt_1 += NUM_LISTS;
        } else {
          sort_tmp[base_cnt_0 + tid] = sort_tmp[i + tid];
          base_cnt_0 += NUM_LISTS;
        }
      }
    }

    for (unsigned long i = 0; i < base_cnt_1; i += NUM_LISTS) {
      if (tid + i < NUM_ELEMENT) {
        sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
      }
    }
    __syncthreads();
  }
}


__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
if (BLOCK_SIZE >= 64) sdata[tid] += sdata[tid + 32];
if (BLOCK_SIZE >= 32) sdata[tid] += sdata[tid + 16];
if (BLOCK_SIZE >= 16) sdata[tid] += sdata[tid + 8];
if (BLOCK_SIZE >= 8) sdata[tid] += sdata[tid + 4];
if (BLOCK_SIZE >= 4) sdata[tid] += sdata[tid + 2];
if (BLOCK_SIZE >= 2) sdata[tid] += sdata[tid + 1];
}
__global__ void sort_block(int *g_idata, int *g_odata, unsigned int n) {
extern __shared__ int sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(BLOCK_SIZE*2) + tid;
unsigned int gridSize = BLOCK_SIZE*2*gridDim.x;
sdata[tid] = 0;
while (i < n) { sdata[tid] += g_idata[i] + g_idata[i+BLOCK_SIZE]; i += gridSize; }
__syncthreads();
if (BLOCK_SIZE >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (BLOCK_SIZE >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (BLOCK_SIZE >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) warpReduce(sdata, tid);
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


__device__ int search_index(unsigned long *const array_tmp, unsigned long val) {
  int left = 0;
  int right = NUM_ELEMENT - 1;
  while (left <= right) {
    int middle = (right + left) / 2;
    if (array_tmp[middle] > val) {
      right = middle - 1;

    } else if (array_tmp[middle] < val) {
      left = middle + 1;
    } else {
      return middle;
    }
  }
  return -1;
  //二分法找元素index
}
template <typename S>
__global__ void sort_index(S *sortarray, unsigned long *const array_tmp,
                           unsigned long *const data) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      data[tid + i] = search_index(array_tmp, sortarray[i + tid].key);
    }
  }
  __syncthreads();
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      array_tmp[data[tid + i]] = tid + i;
    }
  }
  __syncthreads();
}

template <typename S>
__global__ void sort_struct(unsigned long *const array_tmp, S *sortarray,
                            S *struct_tmp) {
  const unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  const unsigned int tid = ix + iy * (gridDim.x * blockDim.x);
  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      struct_tmp[tid + i] = sortarray[array_tmp[tid + i]];
    }
  }
  __syncthreads();

  for (int i = 0; i < NUM_ELEMENT; i += NUM_LISTS) {
    if (tid + i < NUM_ELEMENT) {
      sortarray[tid + i] = struct_tmp[tid + i];
    }
  }
  __syncthreads();
}

template <typename S>
__device__ int search(S *nums, unsigned long val) {
  for (int i = 0; i < NUM_ELEMENT; i++) {
    if (nums[i].key == val) {
      return i;
    }
  }
  return -1;
}
typedef struct SORTSTRUCT {
  unsigned long key;
} sorta;

sorta sortarray[NUM_ELEMENT];  //定义为全局变量避免堆栈溢出

int main(void) {
  for (unsigned long i = 0; i < NUM_ELEMENT; i++) {
    sortarray[i].key = i;
    // sortarray[i].key = i%35;//key值相等的情况
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    c_swap(sortarray[rand() % 7].key, sortarray[i].key);
  }
  sorta *gpu_sortarray;
  sorta *struct_tmp;
  unsigned long *array_tmp;
  unsigned long *array_tmp_2;
  unsigned long *sort_tmp;
  unsigned long *min_array;
  unsigned long *min_value;
  hipMalloc((sorta **)&gpu_sortarray, sizeof(sorta) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&array_tmp_2, sizeof(unsigned long) * NUM_ELEMENT);
  hipMalloc((void **)&sort_tmp, sizeof(unsigned long) * NUM_LISTS);
  hipMalloc((void **)&min_array, sizeof(unsigned long) * GRID_SIZE);
  hipMalloc((sorta **)&struct_tmp, sizeof(sorta) * NUM_ELEMENT);

  radix_sort<<<GRID_SIZE, BLOCK_SIZE>>>(gpu_sortarray, array_tmp, sort_tmp);

  for (int i = 0; i < NUM_ELEMENT; i++) {
    min_value = 0xFFFFFFF;
    merge_block<<<GRID_SIZE, BLOCK_SIZE>>>(sort_tmp, min_array);
    merge_final<<<1, BLOCK_SIZE>>>(min_array, min_value);
    for (int j = 0; j < NUM_LISTS; j++) {
      if (self_data[j] == min_value) {
        sortarray_tmp[i] = min_value;
        index[j] = index[j] + 1;
      }
      if (j + index[j] * NUM_LISTS < NUM_ELEMENT) {
        sort_tmp[j] = array_tmp[j + index[j] * NUM_LISTS];
      } else {
        sort_tmp[j] = 0xFFFFFFFF;
      }
    }
  }
  sort_index<<<GRID_SIZE, BLOCK_SIZE>>>(gpu_sortarray, array_tmp, array_tmp_2);
  sort_struct<<<GRID_SIZE, BLOCK_SIZE>>>(array_tmp, gpu_sortarray, struct_tmp);
  ///////////////////////////////////////////
  hipMemcpy(gpu_sortarray, sortarray, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyHostToDevice);


  hipEvent_t start, stop;  //定义事件
  hipEventCreate(&start);  //起始时间
  hipEventCreate(&stop);   //结束时间

  hipEventRecord(start, 0);  //记录起始时间
  



  hipDeviceSynchronize();
  hipEventRecord(stop, 0);  //执行完代码，记录结束时间

  hipEventSynchronize(stop);
  hipMemcpy(sortarray, array_tmp, sizeof(sorta) * NUM_ELEMENT,
             hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  hipFree(gpu_srcData);
  hipFree(array_tmp);
  hipFree(gpu_sortarray);
  hipFree(struct_tmp);

  printf("CUDA error: %s\n", hipGetErrorString(error));

  int result = 0;
  for (int i = 0; i < NUM_ELEMENT - 1; i++) {
    if (sortarray[i].key > sortarray[i + 1].key) {
      result++;
    }
    // printf("%ld\n",sortarray[i].key);
  }

  for (int i = 0; i < NUM_ELEMENT; i++) {
    printf("%ld\n", sortarray[i].key);
  }

  printf("%ld\n", NUM_ELEMENT);
  printf("%d\n", result);
  if (result == 0) {
    printf("result is true.\n");
  } else {
    printf("result is false.\n");
  }
  float elapsedTime;  //计算总耗时，单位ms
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("%f\n", elapsedTime);
}
